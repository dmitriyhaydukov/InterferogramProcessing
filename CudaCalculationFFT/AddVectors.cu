#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int   Add2(int * aa, int * bb, int * cc, int dd) {
	hipError_t cudaStatus = addWithCuda(cc, aa, bb, dd);
	cudaStatus = hipDeviceReset();
	return 0;
};


/////////////////////////////////////////////////   
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);
	cudaStatus = hipDeviceSynchronize();
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_c); hipFree(dev_a); hipFree(dev_b);
	return cudaStatus;
}