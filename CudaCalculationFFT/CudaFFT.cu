#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <conio.h>
#include <hipfft/hipfft.h>
#include <locale.h>
#include "CudaCalculationFFT.h"

// ���������������� ������� c ������������� ����������� ������
//
// inputMatrix - ��������� �� �������� �������
// outputMatrix - ��������� �� ������� ���������
// width - ������ �������� ������� (��� �� ������ �������-����������)
// height - ������ �������� ������� (��� �� ������ �������-����������)
//
__global__ void transposeMatrixFast(Complex* inputMatrix, Complex* outputMatrix, int width, int height)
{
	__shared__ Complex temp[BLOCK_DIM][BLOCK_DIM];
	int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	if ((xIndex < width) && (yIndex < height))
	{
		// �������� ������ �������� ������ �������� �������  
		int idx = yIndex * width + xIndex;
		//�������� �������� �������� �������
		temp[threadIdx.y][threadIdx.x].x = inputMatrix[idx].x;
		temp[threadIdx.y][threadIdx.x].y = inputMatrix[idx].y;
	}
	//�������������� ��� ���� � �����
	__syncthreads(); // ���������� �� �� �������?!

	xIndex = blockIdx.y * blockDim.y + threadIdx.x;
	yIndex = blockIdx.x * blockDim.x + threadIdx.y;
	if ((xIndex < height) && (yIndex < width))
	{
		// �������� ������ �������� ������ �������� �������  
		int idx = yIndex * height + xIndex;
		//�������� �������� �������� �������
		outputMatrix[idx].x = temp[threadIdx.x][threadIdx.y].x;
		outputMatrix[idx].y = temp[threadIdx.x][threadIdx.y].y;
	}
}

void CalcFFT(Complex* inputMatrix, Complex* outputMatrix, int width, int height)
{
	int matrixSize = width * height;
			
	Complex* devInputMatrix;

	hipHostGetDevicePointer(&devInputMatrix, inputMatrix, 0);

	setlocale(LC_ALL, "Russian");
	
	// ������ ���� ��� ��� �� ��������
	hipfftHandle planY;
	hipfftPlan1d(&planY, height, HIPFFT_Z2Z, width);
	// ������ ���� ��� ��� �� �������
	hipfftHandle planX;
	hipfftPlan1d(&planX, width, HIPFFT_Z2Z, height);
	
	// ��������� ���� - ������������ ������ �� ��������
	dim3 gridSize = dim3(width / BLOCK_DIM, height / BLOCK_DIM, 1);
	dim3 blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);

	// ������������� ������� (��� �� ��������)
	transposeMatrixFast << < gridSize, blockSize >> >(devInputMatrix, devInputMatrix, width, height);
	hipDeviceSynchronize();
	hipfftExecZ2Z(planY, (Complex *)devInputMatrix, (Complex *)devInputMatrix, HIPFFT_BACKWARD);
	hipDeviceSynchronize();

	// ��������� ���� - ������������ ������ �� �������	
	gridSize = dim3(height / BLOCK_DIM, width / BLOCK_DIM, 1);
	blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	// ������������� ������� (��� �� �������)
	transposeMatrixFast << < gridSize, blockSize >> >(devInputMatrix, devInputMatrix, height, width);
	hipDeviceSynchronize();
	hipfftExecZ2Z(planX, (Complex *)devInputMatrix, (Complex *)devInputMatrix, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	
	printf("CUDA RESULT:\n");
	for (int i = 0; i < matrixSize; i++) { if (i % width == 0) printf("\n"); printf("(%4.1f,%4.1f)", inputMatrix[i].x, inputMatrix[i].y); }
	
	hipFree(inputMatrix);
	hipHostFree(devInputMatrix);
		
	getch();
}