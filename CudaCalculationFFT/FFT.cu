#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <conio.h>
#include <hipfft/hipfft.h>
#include <locale.h>
#include "CudaCalculationFFT.h"


__global__ void transposeMatrixFast(Complex* inputMatrix, Complex* outputMatrix, int width, int height)
{
	__shared__ Complex temp[BLOCK_DIM][BLOCK_DIM];
	int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	if ((xIndex < width) && (yIndex < height))
	{
		// �������� ������ �������� ������ �������� �������  
		int idx = yIndex * width + xIndex;
		//�������� �������� �������� �������
		temp[threadIdx.y][threadIdx.x].x = inputMatrix[idx].x;
		temp[threadIdx.y][threadIdx.x].y = inputMatrix[idx].y;
	}
	//�������������� ��� ���� � �����
	__syncthreads(); // ���������� �� �� �������?!

	xIndex = blockIdx.y * blockDim.y + threadIdx.x;
	yIndex = blockIdx.x * blockDim.x + threadIdx.y;
	if ((xIndex < height) && (yIndex < width))
	{
		// �������� ������ �������� ������ �������� �������  
		int idx = yIndex * height + xIndex;
		//�������� �������� �������� �������
		outputMatrix[idx].x = temp[threadIdx.x][threadIdx.y].x;
		outputMatrix[idx].y = temp[threadIdx.x][threadIdx.y].y;
	}
	
}

void CalcFFT(Complex* inputMatrix, Complex* outputMatrix, int width, int height)
{
	hipEvent_t startCopyInput, stopCopyInput;
	float elapsedTimeCopyInput = 0;

	hipEvent_t startCopyOutput, stopCopyOutput;
	float elapsedTimeCopyOutput = 0;
	
	hipEvent_t startCalcFFT, stopCalcFFT;
	float elapsedTimeCalcFFT = 0;

	hipEvent_t startCreatePlan, stopCreatePlan;
	float elapsedTimeCreatePlan = 0;

	int matrixSize = width * height;
	int byteSize = matrixSize * sizeof(Complex);
	
	Complex* devInputMatrix;
	Complex* input;
	
	Complex* transposedMatrix;
	Complex* devTransposedMatrix;

	hipHostAlloc((void**)&input, byteSize, hipHostMallocMapped || hipHostMallocWriteCombined);
	hipHostAlloc((void**)&transposedMatrix, byteSize, hipHostMallocMapped || hipHostMallocWriteCombined);

	hipEventCreate(&startCopyInput);
	hipEventCreate(&stopCopyInput);
	hipEventRecord(startCopyInput, 0);
	
	hipMemcpy(input, inputMatrix, byteSize, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	
	hipEventRecord(stopCopyInput, 0);
	hipEventSynchronize(stopCopyInput);
	hipEventElapsedTime(&elapsedTimeCopyInput, startCopyInput, stopCopyInput);
	printf("\n\n (DLL) Copy input array time: %5.3f ���. \n\n", elapsedTimeCopyInput / 1000);

	hipHostGetDevicePointer(&devInputMatrix, input, 0);
	hipHostGetDevicePointer(&devTransposedMatrix, transposedMatrix, 0);
		
	setlocale(LC_ALL, "Russian");
	
	hipEventCreate(&startCreatePlan);
	hipEventCreate(&stopCreatePlan);
	hipEventRecord(startCreatePlan, 0);
	
	// ������ ���� ��� ��� �� ��������
	hipfftHandle planY;
	hipfftPlan1d(&planY, height, HIPFFT_Z2Z, width);
	// ������ ���� ��� ��� �� �������
	hipfftHandle planX;
	hipfftPlan1d(&planX, width, HIPFFT_Z2Z, height);
		
	hipEventRecord(stopCreatePlan, 0);
	hipEventSynchronize(stopCreatePlan);
	hipEventElapsedTime(&elapsedTimeCreatePlan, startCreatePlan, stopCreatePlan);
	printf("\n\n (DLL) Plan creation FFT time: %5.3f ���. \n\n", elapsedTimeCreatePlan / 1000);
	
	hipEventCreate(&startCalcFFT);
	hipEventCreate(&stopCalcFFT);
	hipEventRecord(startCalcFFT, 0);

	// ��������� ���� - ������������ ������ �� ��������
	dim3 gridSize = dim3(width / BLOCK_DIM, height / BLOCK_DIM, 1);
	dim3 blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	
	// ������������� ������� (��� �� ��������)
	//transposeMatrixFast << < gridSize, blockSize >> >(devInputMatrix, devInputMatrix, width, height);
	//hipDeviceSynchronize();
	//hipfftExecZ2Z(planY, (Complex *)devInputMatrix, (Complex *)devInputMatrix, HIPFFT_BACKWARD);
	//hipDeviceSynchronize();
	
	// ������������� ������� (��� �� ��������)
	transposeMatrixFast << < gridSize, blockSize >> >(devInputMatrix, devTransposedMatrix, width, height);
	hipDeviceSynchronize();
	hipfftExecZ2Z(planY, (Complex *)devTransposedMatrix, (Complex *)devTransposedMatrix, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	

	// ��������� ���� - ������������ ������ �� �������	
	gridSize = dim3(height / BLOCK_DIM, width / BLOCK_DIM, 1);
	blockSize = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	
	// ������������� ������� (��� �� �������)
	transposeMatrixFast << < gridSize, blockSize >> >(devTransposedMatrix, devInputMatrix, height, width);
	hipDeviceSynchronize();
	
	hipfftExecZ2Z(planX, (Complex *)devInputMatrix, (Complex *)devInputMatrix, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	
	hipEventRecord(stopCalcFFT, 0);
	hipEventSynchronize(stopCalcFFT);
	hipEventElapsedTime(&elapsedTimeCalcFFT, startCalcFFT, stopCalcFFT);
	printf("\n\n (DLL) CUDA FFT Calculation time: %5.3f ���. \n\n", elapsedTimeCalcFFT / 1000);

	hipEventCreate(&startCopyOutput);
	hipEventCreate(&stopCopyOutput);
	hipEventRecord(startCopyOutput, 0);
			
	hipMemcpy(outputMatrix, input, byteSize, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	//hipMemcpy(outputMatrix, inputMatrix, byteSize, hipMemcpyDeviceToHost);
	//hipDeviceSynchronize();
	

	hipEventRecord(stopCopyOutput, 0);
	hipEventSynchronize(stopCopyOutput);
	hipEventElapsedTime(&elapsedTimeCopyOutput, startCopyOutput, stopCopyOutput);
	printf("\n\n (DLL) Copy to output array time: %5.3f ���. \n\n", elapsedTimeCopyOutput / 1000);
	
	//printf("\nCUDA OUTPUT\n");
	//printf("(%4.1f,%4.1f)", input[0].x, input[0].y);

	/*
	for (int i = 0; i < matrixSize; i++) { 
		if (i % width == 0) printf("\n"); 
		printf("(%4.1f,%4.1f)", inputMatrix[i].x, inputMatrix[i].y);
	}
	*/
	
	getch();
}
